
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <assert.h>

__global__ void partial_sum(long num, double *out) {
        int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y, index = x + y*blockDim.x*gridDim.x;
        double sum = 0.0;
        double cur = index*num + 1;
        for (long i = 0; i < num; ++i) {
                sum += 1.0/cur;
                cur += 1.0;
        }
        out[index] = sum;
}

__global__ void add_harmonics(double start, double *partials, long num) {
        partials[num] = start;
        for (long i = 0; i < num; ++i) {
                partials[num] += partials[i];
        }
}

int main(int argc, char **argv) {
        if (argc < 2) {
                printf("usage:\n%s <N_ITERATIONS>\n", *argv);
                return -1;
        }
        dim3 block(32, 8);
        long threads_per_block = block.x * block.y, block_w = 6, block_h = 2, blocks = block_w * block_h, threads = threads_per_block*blocks;
        long terms = (long)strtod(argv[1], 0), iterations_per_thread = terms/threads, iterations_left = terms%threads;
        long bytes = (threads+1) * sizeof(double); // last elem is sum of all
        dim3 grid(block_w, block_h);
        double *partials, harmonics = 0.0;
        for (long i = terms-iterations_left; i <= terms; ++i) {
                harmonics += 1.0/i;
        }
        hipMalloc(&partials, bytes);
        partial_sum <<<grid, block>>> (iterations_per_thread, partials);
        hipDeviceSynchronize();
        add_harmonics <<<1, 1>>> (harmonics, partials, threads); // we want to compute the sum of partial sums on the device
        hipMemcpy(&harmonics, partials+threads, sizeof(double), hipMemcpyDeviceToHost);
        hipFree(partials);
        double gamma = harmonics - log(terms);
        printf("%.17f\n", gamma);
        return 0;
}
